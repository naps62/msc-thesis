#include "FVL/CFVProfile.h"

namespace FVL {

	FVLog CFVProfile::stream(FV_PROFILE);

	CFVProfile::CFVProfile(string msg) {
		init(msg, FV_PROFILE);
	}

	CFVProfile::~CFVProfile() {
		hipEventDestroy(start_t);
		hipEventDestroy(stop_t);
	}

	void CFVProfile::start() {
		hipEventRecord(start_t, 0);
	}

	void CFVProfile::stop() {
		hipEventRecord(stop_t, 0);
		hipEventSynchronize(stop_t);
		hipEventElapsedTime(&time, start_t, stop_t);
		stream << "EVENT: " << msg << ": ";
		stream << time << endl;
	}

	float CFVProfile::getTime() {
		return time;
	}

	void CFVProfile::init(string msg, string) {
		this->msg = msg;
		hipEventCreate(&start_t);
		hipEventCreate(&stop_t);
	}
}

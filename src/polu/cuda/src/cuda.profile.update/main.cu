#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <tk/stopwatch.hpp>
#include <hip/hip_runtime.h>

namespace profile {
	tk::Stopwatch *s;
	//PROFILE_COUNTER_CLASS * PROFILE_COUNTER_NAME;

	#define COUNT 7
	float up[COUNT];
	int count[COUNT];

	hipEvent_t start_t, stop_t;

	void init() {
		s = new tk::Stopwatch();

		for(unsigned int i = 0; i < COUNT; ++i)
			up[i] = count[i] = 0;

		hipEventCreate(&start_t);
		hipEventCreate(&stop_t);
	}

	inline void output(std::ostream& out) {
		for(unsigned int i = 0; i < COUNT; ++i) {
			out << ((double)up[i]/(double)count[i]);
			if (i != COUNT - 1)
				out << ';';
		}
		out << endl;
	}

	void cleanup() {
		delete s;

		hipEventDestroy(start_t);
		hipEventDestroy(stop_t);
	}

	inline void time_up(int x) {
		//up[x] += s->last().microseconds();
		float elapsed;
		hipEventElapsedTime(&elapsed, start_t, stop_t);
		up[x] += elapsed;
		count[x]++;
	}

	inline void cuda_start() {
		hipEventRecord(start_t);
	}

	inline void cuda_stop() {
		hipEventRecord(stop_t);
		hipEventSynchronize(stop_t);
	}
}

#define NUM_ITERATIONS  200

#define PROFILE_COUNTER              profile::s
#define PROFILE_INIT()               profile::init()
#define PROFILE_OUTPUT()             profile::output(cout)
#define PROFILE_CLEANUP()            profile::cleanup()
#define PROFILE_START() profile::cuda_start()
#define PROFILE_STOP()  profile::cuda_stop()

#define PROFILE_RETRIEVE_UP(x) profile::time_up(x)


#include "FVL/CFVMesh2D.h"
#include "FVL/CFVArray.h"
#include "FVL/FVXMLReader.h"
#include "FVL/FVXMLWriter.h"
#include "FVio.h"
#include "FVL/FVParameters.h"
using namespace std;

#define  _CUDA 1
#include <hip/hip_runtime.h>
#include "kernels_cuda.cuh"

#define BLOCK_SIZE_UPDATE			512
#define GRID_SIZE(elems, threads)	((int) std::ceil((double)elems/threads))

#define _CUDA_ONLY      if (_CUDA)
#define _NO_CUDA_ONLY   if (!_CUDA)

/**
 * Parameters struct passed via xml file
 */
struct Parameters {
	string mesh_file;
	string velocity_file;
	string initial_file;
	string output_file;
	double final_time;
	double anim_time;
	int anim_jump;
	double dirichlet;
	double CFL;

	public:
	// Constructor receives parameter file
	Parameters(string parameters_filename) {
		FVL::FVParameters para(parameters_filename);

		this->mesh_file		= para.getString("MeshName");
		this->velocity_file	= para.getString("VelocityFile");
		this->initial_file	= para.getString("PoluInitFile");
		this->output_file	= para.getString("OutputFile");
		this->final_time	= para.getDouble("FinalTime");
		this->anim_time		= para.getDouble("AnimTimeStep");
		this->anim_jump		= para.getInteger("NbJump");
		this->dirichlet		= para.getDouble("DirichletCondition");
		this->CFL			= para.getDouble("CFL");
	}
};

int main(int argc, char **argv) {
	

	// var declaration
	int i = 0;
	double h, t, dt, v_max = 0;
	string name;

	// read params
	string param_filename;
	if (argc != 2) {
		param_filename = "param.xml";
	} else
		param_filename = argv[1];

	Parameters data(param_filename);

	// read mesh
	FVL::CFVMesh2D           mesh(data.mesh_file);
	FVL::CFVArray<double>    polution(mesh.num_cells);		// polution arrays
	FVL::CFVArray<double>    flux(mesh.num_edges);			// flux array
	FVL::CFVPoints2D<double> velocities(mesh.num_cells);	// velocities by cell (to calc vs array)
	FVL::CFVArray<double>    vs(mesh.num_edges);			// velocities by edge
	FVL::CFVMat<double> dummy(MAX_EDGES_PER_CELL, 1, mesh.num_cells);

	// read other input files
	FVL::FVXMLReader velocity_reader(data.velocity_file);
	FVL::FVXMLReader polu_ini_reader(data.initial_file);
	velocity_reader.getPoints2D(velocities, t, name);
	polu_ini_reader.getVec(polution, t, name);
	polu_ini_reader.close();
	velocity_reader.close();

	// compute velocity vector
	// TODO: Convert to CUDA
	#ifdef _CUDA
		kernel_compute_edge_velocities(mesh, velocities, vs, v_max);
		h = kernel_compute_mesh_parameter(mesh);
	#else
		cpu_compute_edge_velocities(mesh, velocities, vs, v_max);
		h = cpu_compute_mesh_parameter(mesh);
	#endif

	dt	= 1.0 / v_max * h;

	#ifdef _CUDA
		// saves whole mesh to CUDA memory
		mesh.cuda_malloc();
		polution.cuda_malloc();
		flux.cuda_malloc();
		vs.cuda_malloc();
		dummy.cuda_malloc();

		// data copy
		hipStream_t stream;
		hipStreamCreate(&stream);

		mesh.cuda_save(stream);
		polution.cuda_save(stream);
		vs.cuda_save(stream);
	
		// block and grid sizes for each kernel
		dim3 grid_update(GRID_SIZE(mesh.num_cells, 512), 1, 1);
		dim3 block_update(512, 1, 1);

		dim3 grid_update2(GRID_SIZE(mesh.num_cells, 768), 1, 1);
		dim3 block_update2(768, 1, 1);
	#endif

	PROFILE_INIT();
	//
	// main loop start
	//
	for(unsigned int i = 0; i < NUM_ITERATIONS; ++i) {		
		PROFILE_START();
		kernel_update1<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt);
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(0);

		PROFILE_START();
		kernel_update2<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt);
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(1);

		PROFILE_START();
		kernel_update3<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt);
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(2);

		PROFILE_START();
		kernel_update4<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt);
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(3);

		PROFILE_START();
		kernel_update5<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt, dummy.cuda_get());
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(4);

		PROFILE_START();
		kernel_update6<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt, dummy.cuda_get());
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(5);

		// same kernel, different block size
		PROFILE_START();
		kernel_update6<<< grid_update2, block_update2 >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt, dummy.cuda_get());
		PROFILE_STOP();
		PROFILE_RETRIEVE_UP(6);
	}

	#ifdef _CUDA
		vs.cuda_free();
		polution.cuda_free();
		flux.cuda_free();

		_DEBUG cudaCheckError(string("final check"));
	#endif

	// PROFILE ZONE --- measure postprocessing time
	
	PROFILE_STOP();
	PROFILE_OUTPUT();
	PROFILE_CLEANUP();
}


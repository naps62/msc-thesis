#include "hip/hip_runtime.h"
#include "kernels_cuda.cuh"

__host__ void cudaSafe(hipError_t error, const string msg) {
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << error << endl;
		exit(-1);
	}
}

__host__ void cudaCheckError(const string msg) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << hipGetErrorString(error) << endl;
		exit(-1);
	}
}

// TODO: convert to cudaa
__host__  double kernel_compute_mesh_parameter(CFVMesh2D &mesh) {
	double h;
	double S;

	h = 1.e20;
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {
		S = mesh.cell_areas[cell];

		for(unsigned int edge = 0; edge < mesh.cell_edges_count[cell]; ++edge) {
			double length = mesh.edge_lengths[edge];
			if (h * length > S)
				h = S / length;
		}
	}

	return h;
}

__host__ void kernel_compute_edge_velocities(CFVMesh2D &mesh, CFVPoints2D<double> &velocities, CFVArray<double> &vs, double &v_max) {
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		unsigned int left	= mesh.edge_left_cells[i];
		unsigned int right	= mesh.edge_right_cells[i];

		if (right == NO_RIGHT_CELL)
			right = left;

		double v	= ((velocities.x[left] + velocities.x[right]) * 0.5 * mesh.edge_normals.x[i])
					+ ((velocities.y[left] + velocities.y[right]) * 0.5 * mesh.edge_normals.y[i]);

		vs[i] = v;

		if (abs(v) > v_max || i == 0) {
			v_max = abs(v);
		}
	}
}

__host__ void kernel_compute_length_area_ratio(CFVMesh2D &mesh, CFVMat<double> &length_area_ratio) {
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {

		unsigned int edge_limit = mesh.cell_edges_count[cell];
		for(unsigned int edge_i = 0; edge_i < edge_limit; ++edge_i) {
			unsigned int edge = mesh.cell_edges.elem(edge_i, 0, cell);

			length_area_ratio.elem(edge_i, 0, cell) = mesh.edge_lengths[edge] / mesh.cell_areas[cell];
		}
	}
}

__global__
void kernel_compute_flux(CFVMesh2D_cuda *mesh, double *polution, double *velocity, double *flux, double dc) {
	// thread id = edge index
	unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (edge >= mesh->num_edges) return;

	// velocity of current edge
	double v = velocity[edge];

	unsigned int i_left = mesh->edge_left_cells[edge];
	unsigned int i_right = mesh->edge_right_cells[edge];

	double p_left, p_right;

	p_left	= polution[i_left];
	

	if (i_right != NO_RIGHT_CELL) {
		p_right	 	= polution[i_right];
	} else {
		p_right		= dc;
	}

	/*if (v < 0)
		flux[edge] = v * polution[ mesh->edge_left_cells[edge] ];
	else
		flux[edge] = v * ((mesh->edge_right_cells[edge] == NO_RIGHT_CELL) ? dc : polution[ mesh->edge_right_cells[edge] ]);*/
	if (v >= 0)
		flux[edge] = v * p_left;
	else
		flux[edge] = v * p_right;
}

/**
 * Optimization 1 - optimized flux array access
 */
__global__
void kernel_compute_flux_optim(CFVMesh2D_cuda *mesh, double *polution, double *velocity, double *flux, double dc) {
	unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;

	if (edge >= mesh->num_edges) return;

	unsigned int i_left  = mesh->edge_left_cells[edge];
	unsigned int i_right = mesh->edge_right_cells[edge];

	double p_left, p_right;
	p_left = polution[i_left];

	if (i_right != NO_RIGHT_CELL)
		p_right = polution[i_right];
	else
		p_right = dc;

	double res = velocity[edge];
	if (res >= 0)
		res *= p_left;
	else
		res *= p_right;

	flux[edge] = res;
}

__global__
void kernel_update(CFVMesh2D_cuda *mesh, double *polution, double *flux, double dt) {

	// thread id (cell index)
	unsigned int cell = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (cell >= mesh->num_cells) return;

	// define start and end of neighbor edges
	unsigned int edge_limit = mesh->cell_edges_count[cell];

	// get current polution value for this cell
	double new_polution	= polution[cell];

	// for each edge of this cell
	for(unsigned int edge_i = 0; edge_i < edge_limit; ++edge_i) {
		unsigned int edge = mesh->cell_edges[edge_i][cell];
		// if this cell is at the left of the edge

		// amount of polution transfered through the edge
		double aux = dt * flux[edge] *
			mesh->edge_lengths[edge] /
			mesh->cell_areas[cell];

		// if this cell is on the left or the right of the edge
		if (mesh->edge_left_cells[edge] == cell) {
			new_polution -= aux;
		} else {
			new_polution += aux;
		}
	}

	polution[cell] = new_polution;
}

__global__
void kernel_update2(CFVMesh2D_cuda *mesh, double *polution, double *flux, double dt, double **length_area_ratio) {

	// thread id (cell index)
	unsigned int cell = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (cell >= mesh->num_cells) return;

	// define start and end of neighbor edges
	unsigned int edge_limit = mesh->cell_edges_count[cell];

	// get current polution value for this cell
	double new_polution	= polution[cell];

	// for each edge of this cell
	for(unsigned int edge_i = 0; edge_i < edge_limit; ++edge_i) {
		unsigned int edge = mesh->cell_edges[edge_i][cell];
		// if this cell is at the left of the edge

		// amount of polution transfered through the edge
		double aux = dt * flux[edge] * length_area_ratio[edge_i][cell];

		// if this cell is on the left or the right of the edge
		if (mesh->edge_left_cells[edge] == cell) {
			new_polution -= aux;
		} else {
			new_polution += aux;
		}
	}

	polution[cell] = new_polution;
}

/**
 * Optimization 5 -- added syncthreads (something is wrong with this)
 */
__global__
void kernel_update_optim(CFVMesh2D_cuda *mesh, double *polution, double *flux, double dt, double **length_area_ratio) {

	// thread id (cell index)
	unsigned int cell = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (cell >= mesh->num_cells) return;

	// get current polution value for this cell
	double new_polution	= polution[cell];

	// for each edge of this cell
	for(int edge_i = mesh->cell_edges_count[cell] - 1; edge_i >= 0; --edge_i) {
		unsigned int edge = mesh->cell_edges[edge_i][cell];

		// amount of polution transfered through the edge
		double aux = dt * flux[edge] * length_area_ratio[edge_i][cell];

		// if this cell is on the left or the right of the edge
		// equivalent to:
		   if (mesh->edge_left_cells[edge] == cell) new_polution -= aux;
		   else                                     new_polution += aux;
	}

	__syncthreads();
	polution[cell] = new_polution;
}

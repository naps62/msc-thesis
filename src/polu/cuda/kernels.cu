#include "hip/hip_runtime.h"
#include "kernels.cuh"

__host__ void cudaSafe(hipError_t error, const string msg) {
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << error << endl;
		exit(-1);
	}
}

__host__ void cudaCheckError(const string msg) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << hipGetErrorString(error) << endl;
		exit(-1);
	}
}

// TODO: convert to cudaa
__host__  double kernel_compute_mesh_parameter(CFVMesh2D &mesh) {
	double h;
	double S;

	h = 1.e20;
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {
		S = mesh.cell_areas[cell];

		for(unsigned int edge = 0; edge < mesh.cell_edges_count[cell]; ++edge) {
			double length = mesh.edge_lengths[edge];
			if (h * length > S)
				h = S / length;
		}
	}

	return h;
}

__host__ void kernel_compute_edge_velocities(CFVMesh2D &mesh, CFVPoints2D<double> &velocities, CFVArray<double> &vs, double &v_max) {
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		unsigned int left	= mesh.edge_left_cells[i];
		unsigned int right	= mesh.edge_right_cells[i];

		if (right == NO_RIGHT_CELL)
			right = left;

		double v	= ((velocities.x[left] + velocities.x[right]) * 0.5 * mesh.edge_normals.x[i])
					+ ((velocities.y[left] + velocities.y[right]) * 0.5 * mesh.edge_normals.y[i]);

		vs[i] = v;

		if (abs(v) > v_max || i == 0) {
			v_max = abs(v);
		}
	}
}

__host__ void kernel_compute_length_area_ratio(CFVMesh2D &mesh, CFVMat<double> &length_area_ratio) {
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {

		unsigned int edge_limit = mesh.cell_edges_count[cell];
		for(unsigned int edge_i = 0; edge_i < edge_limit; ++edge_i) {
			unsigned int edge = mesh.cell_edges.elem(edge_i, 0, cell);

			length_area_ratio.elem(edge_i, 0, cell) = mesh.edge_lengths[edge] / mesh.cell_areas[cell];
		}
	}
}


/**
 * Optimization 1 - optimized flux array access
 */
__global__
void kernel_compute_flux_optim(CFVMesh2D_cuda *mesh, double *polution, double *velocity, double *flux, double dc) {
	unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;

	if (edge >= mesh->num_edges) return;

	unsigned int i_left  = mesh->edge_left_cells[edge];
	unsigned int i_right = mesh->edge_right_cells[edge];

	double p_left, p_right;
	p_left = polution[i_left];

	if (i_right != NO_RIGHT_CELL)
		p_right = polution[i_right];
	else
		p_right = dc;

	double res = velocity[edge];
	if (res >= 0)
		res *= p_left;
	else
		res *= p_right;

	flux[edge] = res;
}


/**
 * Optimization 5 -- added syncthreads
 */
__global__
void kernel_update_optim(CFVMesh2D_cuda *mesh, double *polution, double *flux, double dt, double **length_area_ratio) {

	// thread id (cell index)
	unsigned int cell = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (cell >= mesh->num_cells) return;

	// get current polution value for this cell
	double new_polution	= polution[cell];

	// for each edge of this cell
	for(int edge_i = mesh->cell_edges_count[cell] - 1; edge_i >= 0; --edge_i) {
		unsigned int edge = mesh->cell_edges[edge_i][cell];

		// amount of polution transfered through the edge
		double aux = dt * flux[edge] * length_area_ratio[edge_i][cell];

		// if this cell is on the left or the right of the edge
		new_polution += aux * (2*(int)(mesh->edge_left_cells[edge] == cell) - 1);
		// equivalent to:
		//   if (mesh->edge_left_cells) polution -= aux
		//   else                       polution += aux
	}

	__syncthreads();
	polution[cell] = new_polution;
}

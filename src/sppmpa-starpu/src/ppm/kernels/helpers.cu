#include "hip/hip_runtime.h"
#include "ppm/kernels/helpers.cuh"
#include "ppm/ptrfree_hash_grid.h"
#include "ppm/math.h"
#include "ppm/kernels/cu_math.cuh"
#include <limits>
#include <cfloat>

#define QBVH_STACK_SIZE 24

namespace ppm { namespace kernels {

namespace helpers {


__HD__
void tex_map_get_texel(
    const Spectrum* const pixels,
    const unsigned width,
    const unsigned height,
    const int s,
    const int t,
    Spectrum& color) {

  const unsigned u = Mod(s, width);
  const unsigned v = Mod(t, height);

  const Spectrum& pixel = pixels[v * width + u];

  color.r = pixel.r;
  color.g = pixel.g;
  color.b = pixel.b;
}

__HD__
void tex_map_get_color(
    const Spectrum* const pixels,
    const unsigned width,
    const unsigned height,
    const float u,
    const float v,
    Spectrum& color) {

  const float s = u * width  - 0.5f;
  const float t = v * height - 0.5f;

  const int s0 = (int) floor(s);
  const int t0 = (int) floor(t);

  const float ds = s - s0;
  const float dt = t - t0;

  const float ids = 1.f - ds;
  const float idt = 1.f - dt;

  Spectrum c0, c1, c2, c3;
  tex_map_get_texel(pixels, width, height, s0,     t0,     c0);
  tex_map_get_texel(pixels, width, height, s0,     t0 + 1, c1);
  tex_map_get_texel(pixels, width, height, s0 + 1, t0,     c2);
  tex_map_get_texel(pixels, width, height, s0 + 1, t0 + 1, c3);

  const float k0 = ids * idt;
  const float k1 = ids * dt;
  const float k2 = ds  * idt;
  const float k3 = ds  * dt;

  color.r = k0 * c0.r + k1 * c1.r + k2 * c2.r + k3 * c3.r;
  color.g = k0 * c0.g + k1 * c1.g + k2 * c2.g + k3 * c3.g;
  color.b = k0 * c0.b + k1 * c1.b + k2 * c2.b + k3 * c3.b;
}


__HD__
void infinite_light_le (
    Spectrum& le,
    const Vector& dir,
    const InfiniteLight& infinite_light,
    const Spectrum* const infinite_light_map) {

  const float u = 1.f - SphericalPhi(dir)   * INV_TWOPI + infinite_light.shiftU;
  const float v =       SphericalTheta(dir) * INV_PI    + infinite_light.shiftV;

  tex_map_get_color(infinite_light_map, infinite_light.width, infinite_light.height, u, v, le);

  le.r *= infinite_light.gain.r;
  le.g *= infinite_light.gain.g;
  le.b *= infinite_light.gain.b;

}


__HD__
void sky_light_le(
    Spectrum& f,
    const Vector& dir,
    const SkyLight& sky_light) {

  const float theta = SphericalTheta(dir);
  const float phi   = SphericalPhi(dir);

  Spectrum s;
  sky_light_get_sky_spectral_radiance(theta, phi, s, sky_light);

  f.r = sky_light.gain.r * s.r;
  f.g = sky_light.gain.g * s.g;
  f.b = sky_light.gain.b * s.b;
}

__HD__
float sky_light_perez_base(
    const float* const lam,
    const float theta,
    const float gamma) {

  return (1.f + lam[1] * exp(lam[2] / cos(theta))) * (1.f + lam[3] * exp(lam[4] * gamma) + lam[5] * cos(gamma) * cos(gamma));
}

__HD__
void sky_light_chromaticity_to_spectrum(
    const float Y,
    const float x,
    const float y,
    Spectrum& s) {

  float X, Z;

  if (y != 0.f)
    X = (x/y) * Y;
  else
    X = 0.f;

  if (y != 0.f && Y != 0.f)
    Z = (1.f - x - y) / y * Y;
  else
    Z = 0.f;

  // assuming sRGB (D65 illuminant)
  s.r =  3.2410f * X - 1.5374f * Y - 0.4986f * Z;
  s.g = -0.9692f * X + 1.8760f * Y + 0.0416f * Z;
  s.b =  0.0556f * X - 0.2040f * Y + 1.0570f * Z;
}

__HD__
float ri_angle_between(
    const float thetav,
    const float phiv,
    const float theta,
    const float phi) {

  const float cospsi = sinf(thetav) * sin(theta) * cosf(phi - phiv) + cosf(thetav) * cosf(theta);

  if (cospsi >= 1.f)
    return 0.f;
  if (cospsi <= -1.f)
    return M_PI;
  return acosf(cospsi);
}

__HD__
void sky_light_get_sky_spectral_radiance(
    const float theta,
    const float phi,
    Spectrum& spect,
    const SkyLight& sky_light) {

  const float theta_fin = min(theta, (float) ((M_PI * 0.5f) - 0.001f));
  const float gamma     = ri_angle_between(theta, phi, sky_light.theta_s, sky_light.phi_s);

  const float x = sky_light.zenith_x * sky_light_perez_base(sky_light.perez_x, theta_fin, gamma);
  const float y = sky_light.zenith_y * sky_light_perez_base(sky_light.perez_y, theta_fin, gamma);
  const float Y = sky_light.zenith_Y * sky_light_perez_base(sky_light.perez_Y, theta_fin, gamma);

  sky_light_chromaticity_to_spectrum(Y, x, y, spect);
}


__HD__
void sun_light_le(
    Spectrum& le,
    const Vector& dir,
    const SunLight& sun_light) {

  const float cos_theta_max = sun_light.cos_theta_max;
  const Vector sun_dir = sun_light.dir;

  if ((cos_theta_max < 1.f) && (Dot(dir, sun_dir) > cos_theta_max))
    le = sun_light.color;
  else {
    le.r = 0.f;
    le.g = 0.f;
    le.b = 0.f;
  }
}

__HD__
void area_light_le(
    Spectrum& le,
    const Vector& wo,
    const Normal& light_normal,
    const AreaLightParam& mat) {

  const bool bright_side = (Dot(light_normal, wo) > 0.f);

  if (bright_side) {
    le.r = mat.gain.r;
    le.g = mat.gain.g;
    le.b = mat.gain.b;
  } else {
    le.r = 0.f;
    le.g = 0.f;
    le.b = 0.f;
  }
}


__HD__
void concentric_sample_disk(const float u1, const float u2, float *dx, float *dy) {
  float r, theta;
  // Map uniform random numbers to $[-1,1]^2$
  float sx = 2.f * u1 - 1.f;
  float sy = 2.f * u2 - 1.f;
  // Map square to $(r,\theta)$
  // Handle degeneracy at the origin
  if (sx == 0.f && sy == 0.f) {
    *dx = 0.f;
    *dy = 0.f;
    return;
  }
  if (sx >= -sy) {
    if (sx > sy) {
      // Handle first region of disk
      r = sx;
      if (sy > 0.f)
        theta = sy / r;
      else
        theta = 8.f + sy / r;
    } else {
      // Handle second region of disk
      r = sy;
      theta = 2.f - sx / r;
    }
  } else {
    if (sx <= sy) {
      // Handle third region of disk
      r = -sx;
      theta = 4.f - sy / r;
    } else {
      // Handle fourth region of disk
      r = -sy;
      theta = 6.f + sx / r;
    }
  }
  theta *= M_PI / 4.f;
  *dx = r * cosf(theta);
  *dy = r * sinf(theta);
}

__HD__
LightType sample_all_lights(
    const float u,
    const unsigned lights_count,
    const InfiniteLight& infinite_light,
    const SunLight& sun_light,
    const SkyLight& sky_light,
    float& pdf,
    int& light_index,
    const bool skip_infinite_light) {

  if (!skip_infinite_light && (infinite_light.exists || sun_light.exists || sky_light.exists)) {
    unsigned count = lights_count;
    int ilx1 = 0;
    int ilx2 = 0;
    int ilx3 = 0;

    if (infinite_light.exists) ilx1 = count++;
    if (sun_light.exists)      ilx2 = count++;
    if (sky_light.exists)      ilx3 = count++;

    light_index = Floor2UInt(count * u);
    pdf = 1.f / count;

    if      (light_index == ilx1) return ppm::LIGHT_IL_IS;
    else if (light_index == ilx2) return ppm::LIGHT_SUN;
    else if (light_index == ilx3) return ppm::LIGHT_IL_SKY;
    else return ppm::LIGHT_TRIANGLE;

  } else {
    light_index = Min(Floor2UInt(lights_count * u), lights_count - 1);
    pdf = 1.f / lights_count;
    return ppm::LIGHT_TRIANGLE;
  }
}


__HD__
void infinite_light_sample_l(
    const float u0,
    const float u1,
    const float u2,
    const float u3,
    const InfiniteLight& infinite_light,
    const Spectrum* const infinite_light_map,
    const BSphere& bsphere,
    float& pdf, Ray& ray, Spectrum& f) {

  const float rad = bsphere.rad * 1.01f;
  const Point p1 = bsphere.center + rad * UniformSampleSphere(u0, u1);
  const Point p2 = bsphere.center + rad * UniformSampleSphere(u2, u3);

  ray = Ray(p1, Normalize(p2 - p1));

  const Vector to_center = Normalize(bsphere.center - p1);
  const float cos_theta = AbsDot(to_center, ray.d);
  pdf = cos_theta / (4.f * M_PI * M_PI * rad * rad);

  Vector dir = -ray.d;
  infinite_light_le(f, dir, infinite_light, infinite_light_map);
}

__HD__
void sun_light_sample_l(
    const float u0,
    const float u1, const SunLight& sun_light,
    const Point& hit_point,
    float& pdf,
    Ray& shadow_ray,
    Spectrum& f) {

  Vector wi = UniformSampleCone(u0, u1, sun_light.cos_theta_max, sun_light.x, sun_light.y, sun_light.dir);

  shadow_ray.o = hit_point;
  shadow_ray.d = wi;
  shadow_ray.mint = RAY_EPSILON;
  shadow_ray.maxt = FLT_MAX/*std::numeric_limits<float>::max()*/;

  f = sun_light.color;
  pdf = UniformConePdf(sun_light.cos_theta_max);
}

__HD__
void sun_light_sample_l(
    const float u0,
    const float u1,
    const float u2,
    const float u3,
    const SunLight& sun_light,
    const BSphere& bsphere,
    float& pdf,
    Ray& ray,
    Spectrum& f) {

  const float rad = bsphere.rad * 1.01f;

  float d1, d2;
  ConcentricSampleDisk(u0, u1, &d1, &d2);
  const Point p_disk = bsphere.center + rad * (d1 * sun_light.x + d2 * sun_light.y);

  ray = Ray(p_disk + rad * sun_light.dir, -UniformSampleCone(u2, u3, sun_light.cos_theta_max, sun_light.x, sun_light.y, sun_light.dir));
  pdf = UniformConePdf(sun_light.cos_theta_max) / (M_PI * rad * rad);
}

__HD__
void sky_light_sample_l(
    const float u0,
    const float u1,
    const float u2,
    const float u3,
    const SkyLight& sky_light,
    const BSphere& bsphere,
    float& pdf,
    Ray& ray,
    Spectrum& f) {

  const float rad = bsphere.rad * 1.01f;
  const Point p1 = bsphere.center + rad * UniformSampleSphere(u0, u1);
  const Point p2 = bsphere.center + rad * UniformSampleSphere(u2, u3);

  ray = Ray(p1, Normalize(p2 - p1));

  const Vector to_center = Normalize(bsphere.center - p1);
  const float cos_theta = AbsDot(to_center, ray.d);
  pdf = cos_theta / (4.f * M_PI * M_PI * rad * rad);

  const Vector dir = -ray.d;
  sky_light_le(f, dir, sky_light);
}


__HD__
void triangle_light_sample_l(
    const float u0,
    const float u1,
    const float u2,
    const float u3,
    const TriangleLight& light,
    const Mesh* const mesh_descs,
    const Spectrum* const colors,
    float& pdf,
    Ray& ray,
    Spectrum& f) {

  Point orig;
  sample_triangle_light(light, u0, u1, orig);

  const Normal sample_N = light.normal;

  const float z = 1.f - 2.f * u2;
  const float r = sqrtf(Max(0.f, 1.f - z * z));
  const float phi = 2.f * M_PI * u3;
  const float x = r * cosf(phi);
  const float y = r * sinf(phi);

  Vector dir = Vector(x, y, z);
  float RdotN = Dot(dir, sample_N);
  if (RdotN < 0.f) {
    dir *= -1.f;
    RdotN = -RdotN;
  }

  ray = Ray(orig, dir);
  pdf = INV_TWOPI / light.area;
  const Mesh& m = mesh_descs[light.mesh_index];

  f.r = light.gain.r * RdotN;
  f.g = light.gain.g * RdotN;
  f.b = light.gain.b * RdotN;

  if (m.has_colors) {
    const unsigned i = m.colors_offset + light.tri_index;
    f.r *= colors[i].r;
    f.g *= colors[i].g;
    f.b *= colors[i].b;
  }
}

__HD__
void sample_triangle_light(
    const TriangleLight& l,
    const float u0,
    const float u1,
    Point& p) {

  const float su1 = sqrt(u0);
  const float b0 = 1.f - su1;
  const float b1 = u1 * su1;
  const float b2 = 1.f - b0 - b1;

  p.x = b0 * l.v0.x + b1 * l.v1.x + b2 * l.v2.x;
  p.y = b0 * l.v0.y + b1 * l.v1.y + b2 * l.v2.y;
  p.z = b0 * l.v0.z + b1 * l.v1.z + b2 * l.v2.z;
}


__HD__
Ray generate_ray(
    const float sx, const float sy,
    const uint width, const uint height,
    const float u0, const float u1, const float u2, const Camera& camera) {

  Point p(sx, height - sy - 1.f, 0);
  Point orig;

  const float iw = 1.f / (camera.raster_to_camera_matrix[3][0] * p.x
                        + camera.raster_to_camera_matrix[3][1] * p.y
                        + camera.raster_to_camera_matrix[3][2] * p.z
                        + camera.raster_to_camera_matrix[3][3]);
  orig.x = (camera.raster_to_camera_matrix[0][0] * p.x
      + camera.raster_to_camera_matrix[0][1] * p.y
      + camera.raster_to_camera_matrix[0][2] * p.z
      + camera.raster_to_camera_matrix[0][3]) * iw;
  orig.y = (camera.raster_to_camera_matrix[1][0] * p.x
      + camera.raster_to_camera_matrix[1][1] * p.y
      + camera.raster_to_camera_matrix[1][2] * p.z
      + camera.raster_to_camera_matrix[1][3]) * iw;
  orig.z = (camera.raster_to_camera_matrix[2][0] * p.x
      + camera.raster_to_camera_matrix[2][1] * p.y
      + camera.raster_to_camera_matrix[2][2] * p.z
      + camera.raster_to_camera_matrix[2][3]) * iw;

  Vector dir(orig);

  const float hither = camera.hither;
  if (camera.lens_radius > 0.f) {
    // sample point on lens
    float lens_u, lens_v;
    concentric_sample_disk(u1, u2, &lens_u, &lens_v);
    const float lens_radius = camera.lens_radius;
    lens_u *= lens_radius;
    lens_v *= lens_radius;

    // compute point on plane of focus
    const float focal_distance = camera.focal_distance;
    const float dist = focal_distance - hither;
    const float ft = dist / dir.z;
    Point p_focus = orig + dir * ft;

    // update ray for effect on lens
    const float k = dist / focal_distance;
    orig.x += lens_u * k;
    orig.y += lens_v * k;

    dir = p_focus - orig;
  }

  dir = Normalize(dir);

  Point torig;
  const float iw2 = 1.f / ( camera.camera_to_world_matrix[3][0] * orig.x
                      + camera.camera_to_world_matrix[3][1] * orig.y
                      + camera.camera_to_world_matrix[3][2] * orig.z
                      + camera.camera_to_world_matrix[3][3]);
  torig.x = (camera.camera_to_world_matrix[0][0] * orig.x
      +  camera.camera_to_world_matrix[0][1] * orig.y
      +  camera.camera_to_world_matrix[0][2] * orig.z
      +  camera.camera_to_world_matrix[0][3]) * iw2;
  torig.y = (camera.camera_to_world_matrix[1][0] * orig.x
      +  camera.camera_to_world_matrix[1][1] * orig.y
      +  camera.camera_to_world_matrix[1][2] * orig.z
      +  camera.camera_to_world_matrix[1][3]) * iw2;
  torig.z = (camera.camera_to_world_matrix[2][0] * orig.x
      +  camera.camera_to_world_matrix[2][1] * orig.y
      +  camera.camera_to_world_matrix[2][2] * orig.z
      +  camera.camera_to_world_matrix[2][3]) * iw2;

  Vector tdir;
  tdir.x = camera.camera_to_world_matrix[0][0] * dir.x
       + camera.camera_to_world_matrix[0][1] * dir.y
       + camera.camera_to_world_matrix[0][2] * dir.z;
  tdir.y = camera.camera_to_world_matrix[1][0] * dir.x
       + camera.camera_to_world_matrix[1][1] * dir.y
       + camera.camera_to_world_matrix[1][2] * dir.z;
  tdir.z = camera.camera_to_world_matrix[2][0] * dir.x
       + camera.camera_to_world_matrix[2][1] * dir.y
       + camera.camera_to_world_matrix[2][2] * dir.z;

  return Ray(torig, tdir, RAY_EPSILON, (camera.yon - hither) / dir.z);
}

__HD__
bool get_hit_point_information(
    const PtrFreeScene* const scene,
    Ray& ray,
    const RayHit& hit,
    Point& hit_point,
    Spectrum& surface_color,
    Normal& N,
    Normal& shade_N) {

  hit_point = ray(hit.t);
  const unsigned current_triangle_index = hit.index;

  unsigned current_mesh_index = scene->mesh_ids[current_triangle_index];
  unsigned triangle_index = current_triangle_index - scene->mesh_first_triangle_offset[current_mesh_index];

  const Mesh& m = scene->mesh_descs[current_mesh_index];

  if (m.has_colors) {
    // mesh interpolate color
    mesh_interpolate_color(&scene->colors[m.colors_offset], &scene->triangles[m.tris_offset], triangle_index, hit.b1, hit.b2, surface_color);
  } else {
    surface_color = Spectrum(1.f, 1.f, 1.f);
  }

  // mesh interpolate normal
  mesh_interpolate_normal(&scene->normals[m.verts_offset], &scene->triangles[m.tris_offset], triangle_index, hit.b1, hit.b2, N);

  if (Dot(ray.d, N) > 0.f)
    shade_N = -N;
  else
    shade_N = N;

  return false;

}

__HD__
void mesh_interpolate_color(
    const Spectrum* const colors,
    const Triangle* const triangles,
    const unsigned triangle_index,
    const float b1,
    const float b2,
    Spectrum& C) {

  const Triangle& triangle = triangles[triangle_index];
  const float b0 = 1.f - b1 - b2;

  C.r = b0 * colors[triangle.v[0]].r + b1 * colors[triangle.v[1]].r + b2 * colors[triangle.v[2]].r;
  C.g = b0 * colors[triangle.v[0]].g + b1 * colors[triangle.v[1]].g + b2 * colors[triangle.v[2]].g;
  C.b = b0 * colors[triangle.v[0]].b + b1 * colors[triangle.v[1]].b + b2 * colors[triangle.v[2]].b;
}

__HD__
void mesh_interpolate_normal(
    const Normal* const normals,
    const Triangle* const triangles,
    const unsigned triangle_index,
    const float b1,
    const float b2,
    Normal& N) {

  const Triangle& triangle = triangles[triangle_index];
  const float b0 = 1.f - b1 - b2;

  const Normal& v0 = normals[triangle.v[0]];
  const Normal& v1 = normals[triangle.v[1]];
  const Normal& v2 = normals[triangle.v[2]];

  N.x = b0 * v0.x + b1 * v1.x + b2 * v2.x;
  N.y = b0 * v0.y + b1 * v1.y + b2 * v2.y;
  N.z = b0 * v0.z + b1 * v1.z + b2 * v2.z;

  N = Normalize(N);
}

__HD__
void mesh_interpolate_UV(
    const UV* const uvs,
    const Triangle* const triangles,
    const unsigned triangle_index,
    const float b1,
    const float b2,
    UV& uv) {

  const Triangle& triangle = triangles[triangle_index];
  const float b0 = 1.f - b1 - b2;

  uv.u = b0 * uvs[triangle.v[0]].u + b1 * uvs[triangle.v[1]].u + b2 * uvs[triangle.v[2]].u;
  uv.v = b0 * uvs[triangle.v[0]].v + b1 * uvs[triangle.v[1]].v + b2 * uvs[triangle.v[2]].v;
}

__HD__
void generic_material_sample_f(
    const Material& mat,
    Vector& wo,
    Vector& wi,
    const Normal& N,
    const Normal& shade_N,
    const float u0,
    const float u1,
    const float u2,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  switch (mat.type) {
    case MAT_AREALIGHT:
      break;
    case MAT_MATTE:       helpers::matte_material_sample_f              (mat.param.matte, wo, wi,    shade_N, u0, u1,     pdf, f, specular_bounce); break;
    case MAT_MIRROR:      helpers::mirror_material_sample_f            (mat.param.mirror, wo, wi,    shade_N,             pdf, f, specular_bounce); break;
    case MAT_GLASS:       helpers::glass_material_sample_f              (mat.param.glass, wo, wi, N, shade_N, u0,         pdf, f, specular_bounce); break;
    case MAT_MATTEMIRROR: helpers::matte_mirror_material_sample_f(mat.param.matte_mirror, wo, wi,    shade_N, u0, u1, u2, pdf, f, specular_bounce); break;
    case MAT_METAL:       helpers::metal_material_sample_f              (mat.param.metal, wo, wi,    shade_N, u0, u1,     pdf, f, specular_bounce); break;
    case MAT_MATTEMETAL:  helpers::matte_metal_material_sample_f  (mat.param.matte_metal, wo, wi,    shade_N, u0, u1, u2, pdf, f, specular_bounce); break;
    case MAT_ALLOY:       helpers::alloy_material_sample_f              (mat.param.alloy, wo, wi,    shade_N, u0, u1, u2, pdf, f, specular_bounce); break;
    case MAT_ARCHGLASS:   helpers::arch_glass_material_sample_f    (mat.param.arch_glass, wo, wi, N, shade_N, u0,         pdf, f, specular_bounce); break;
    case MAT_NULL:
      wi = - wo;
      specular_bounce = true;
      pdf = 1.f;
      break;
    default:
      specular_bounce = true;
      pdf = 0.f;
      break;
  }
}

__HD__
void matte_material_sample_f(
    const MatteParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    const float u0,
    const float u1,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  Vector dir = CosineSampleHemisphere(u0, u1);
  pdf = dir.z * INV_PI;

  Vector v1, v2;
  CoordinateSystem((Vector) shade_N, &v1, &v2);

  wi.x = v1.x * dir.x + v2.x * dir.y + shade_N.x * dir.z;
  wi.y = v1.y * dir.x + v2.y * dir.y + shade_N.y * dir.z;
  wi.z = v1.z * dir.x + v2.z * dir.y + shade_N.z * dir.z;

  const float dp = Dot(shade_N, wi);

  if (dp <= 0.0001f) {
    pdf = 0.f;
  } else {
    f.r = mat.kd.r * INV_PI;
    f.g = mat.kd.g * INV_PI;
    f.b = mat.kd.b * INV_PI;
    pdf /= dp;
  }
  specular_bounce = false;
}

__HD__
void mirror_material_sample_f(
    const MirrorParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  const float k = 2.f * Dot(shade_N, wo);
  wi.x = k * shade_N.x - wo.x;
  wi.y = k * shade_N.y - wo.y;
  wi.z = k * shade_N.z - wo.z;

  pdf = 1.f;
  f.r = mat.kr.r;
  f.g = mat.kr.g;
  f.b = mat.kr.b;
  specular_bounce = mat.specular_bounce;
}

__HD__
void glass_material_sample_f(
    const GlassParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& N,
    const Normal& shade_N,
    const float u0,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  const float k = 2.f * Dot(N, wo);
  Vector refl_dir;
  refl_dir.x = k * N.x - wo.x;
  refl_dir.y = k * N.y - wo.y;
  refl_dir.z = k * N.z - wo.z;


  const bool into = (Dot(N, shade_N) > 0.f);
  const float nc = mat.outside_ior;
  const float nt = mat.ior;
  const float nnt = into ? (nc / nt) : (nt / nc);
  const float ddn = -Dot(wo, shade_N);
  const float cos2t = 1.f - nnt * nnt * (1.f - ddn * ddn);

  if (cos2t < 0.f) {
    wi = refl_dir;
    pdf = 1.f;
    f.r = mat.refl.r;
    f.g = mat.refl.g;
    f.b = mat.refl.b;
    specular_bounce = mat.reflection_specular_bounce;
  } else {
    const float kk = (into ? 1.f : -1.f) * (ddn * nnt + sqrt(cos2t));
    Vector nkk = (Vector) N;
    nkk *= kk;

    Vector trans_dir = -nnt * wo - nkk;
    Normalize(trans_dir);

    const float c = 1.f - (into ? -ddn : Dot(trans_dir, N));
    const float R0 = mat.R0;
    const float Re = R0 + (1.f - R0) * c * c * c * c * c;
    const float Tr = 1.f - Re;
    const float P = .25f + .5f * Re;

    if (Tr == 0.f) {
      if (Re == 0.f)
        pdf = 0.f;
      else {
        wi = refl_dir;
        pdf = 1.f;
        f.r = mat.refl.r;
        f.g = mat.refl.g;
        f.b = mat.refl.b;
        specular_bounce = mat.reflection_specular_bounce;
      }

    } else if (Re == 0.f) {
      wi = trans_dir;
      pdf = 1.f;
      f.r = mat.refrct.r;
      f.g = mat.refrct.g;
      f.b = mat.refrct.b;
      specular_bounce = mat.transmission_specular_bounce;

    } else if (u0 < P) {
      wi = refl_dir;
      pdf = P / Re;
      f.r = mat.refl.r;
      f.g = mat.refl.g;
      f.b = mat.refl.b;
      specular_bounce = mat.reflection_specular_bounce;

    } else {
      wi = trans_dir;
      pdf = (1.f - P) / Tr;
      f.r = mat.refrct.r;
      f.g = mat.refrct.g;
      f.b = mat.refrct.b;
      specular_bounce = mat.transmission_specular_bounce;
    }
  }
}

__HD__
void matte_mirror_material_sample_f(
    const MatteMirrorParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    const float u0,
    const float u1,
    const float u2,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  const float tot_filter = mat.tot_filter;
  const float comp = u2 * tot_filter;

  float mpdf;
  if (comp > mat.matte_filter) {
    mirror_material_sample_f(mat.mirror, wo, wi, shade_N, pdf, f, specular_bounce);
    mpdf = mat.mirror_pdf;
  } else {
    matte_material_sample_f(mat.matte, wo, wi, shade_N, u0, u1, pdf, f, specular_bounce);
    mpdf = mat.matte_pdf;
  }

  pdf *= mpdf;
}

__HD__
void metal_material_sample_f(
    const MetalParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    const float u0,
    const float u1,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {
  glossy_reflection(wo, wi, mat.exp, shade_N, u0, u1);

  if (Dot(wi, shade_N) > 0.f) {
    pdf = 1.f;
    f.r = mat.kr.r;
    f.g = mat.kr.g;
    f.b = mat.kr.b;
    specular_bounce = mat.specular_bounce;
  } else {
    pdf = 0.f;
  }
}

__HD__
void matte_metal_material_sample_f(
    const MatteMetalParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    const float u0,
    const float u1,
    const float u2,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {
  const float tot_filter = mat.tot_filter;
  const float comp = u2 * tot_filter;

  float mpdf;
  if (comp > mat.matte_filter) {
    metal_material_sample_f(mat.metal, wo, wi, shade_N, u0, u1, pdf, f, specular_bounce);
    mpdf = mat.metal_pdf;
  } else {
    matte_material_sample_f(mat.matte, wo, wi, shade_N, u0, u1, pdf, f, specular_bounce);
    mpdf = mat.matte_pdf;
  }
  pdf *= mpdf;
}

__HD__
void alloy_material_sample_f(
    const AlloyParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& shade_N,
    const float u0,
    const float u1,
    const float u2,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {

  const float c = 1.f - Dot(wo, shade_N);
  const float R0 = mat.R0;
  const float Re = R0 + (1.f - R0) * c * c * c * c * c;
  const float P = .25f + .5f * Re;

  if (u2 <= P) {
    glossy_reflection(wo, wi, mat.exp, shade_N, u0, u1);
    pdf = P / Re;
    f.r = mat.refl.r * Re;
    f.g = mat.refl.g * Re;
    f.b = mat.refl.b * Re;
    specular_bounce = mat.specular_bounce;
  } else {
    Vector dir = CosineSampleHemisphere(u0, u1);
    pdf = dir.z * INV_PI;

    Vector v1, v2;
    CoordinateSystem((Vector) shade_N, &v1, &v2);

    wi.x = v1.x * dir.x + v2.x * dir.y + shade_N.x * dir.z;
    wi.y = v1.y * dir.x + v2.y * dir.y + shade_N.y * dir.z;
    wi.z = v1.z * dir.x + v2.z * dir.y + shade_N.z * dir.z;

    if (dir.z <= 0.0001f)
      pdf = 0.f;
    else {
      const float iRe = 1.f - Re;
      const float k =  ( 1.f - P) / iRe;
      pdf *= k;
      f.r = mat.diff.r * iRe;
      f.g = mat.diff.g * iRe;
      f.b = mat.diff.b * iRe;
      specular_bounce = false;
    }
  }
}

__HD__
void arch_glass_material_sample_f(
    const ArchGlassParam& mat,
    const Vector& wo,
    Vector& wi,
    const Normal& N,
    const Normal& shade_N,
    const float u0,
    float& pdf,
    Spectrum& f,
    bool& specular_bounce) {


  const bool into = (Dot(N, shade_N) > 0.f);

  if (!into) {
    wi = -wo;
    pdf = 1.f;
    f.r = mat.refrct.r;
    f.g = mat.refrct.g;
    f.b = mat.refrct.b;
    specular_bounce = mat.transmission_specular_bounce;
  } else {
    const float comp = u0 * mat.tot_filter;

    if (comp > mat.trans_filter) {
      const float k = 2.f * Dot(N, wo);
      wi.x = k * N.x - wo.x;
      wi.y = k * N.y - wo.y;
      wi.z = k * N.z - wo.z;
      pdf = mat.refl_pdf;

      f.r = mat.refl.r;
      f.g = mat.refl.g;
      f.b = mat.refl.b;
      specular_bounce = mat.reflection_specular_bounce;
    } else {
      wi = -wo;
      pdf = mat.trans_pdf;
      f.r = mat.refrct.r;
      f.g = mat.refrct.g;
      f.b = mat.refrct.b;
      specular_bounce = mat.transmission_specular_bounce;
    }
  }
}

__HD__
void glossy_reflection(
    const Vector& wo,
    Vector& wi,
    const float exponent,
    const Normal& shade_N,
    const float u0,
    const float u1) {

  const float phi = 2.f * M_PI * u0;
  const float cos_theta = powf(1.f - u1, exponent);
  const float sin_theta = sqrtf(Max(0.f, 1.f - cos_theta * cos_theta));
  const float x = cosf(phi) * sin_theta;
  const float y = sinf(phi) * sin_theta;
  const float z = cos_theta;

  const Vector dir = -wo;
  const float dp = Dot(shade_N, dir);
  const Vector w = dir - (2.f * dp) * Vector(shade_N);

  Vector u;
  if (fabsf(shade_N.x) > .1f) {
    const Vector a(0.f, 1.f, 0.f);
    u = Cross(a, w);
  } else {
    const Vector a(1.f, 0.f, 0.f);
    u = Cross(a, w);
  }
  u = Normalize(u);
  Vector v = Cross(w, u);

  wi = x * u + y * v + z * w;
}


__HD__ void matte_f(
    const MatteParam& mat,
    Spectrum& f) {

  f.r = mat.kd.r * INV_PI;
  f.g = mat.kd.g * INV_PI;
  f.b = mat.kd.b * INV_PI;
}

__HD__ void matte_mirror_f(
    const MatteMirrorParam& mat,
    Spectrum& f) {
  matte_f(mat.matte, f);
  f *= mat.matte_pdf;
}

__HD__ void matte_metal_f(
    const MatteMetalParam& mat,
    Spectrum& f) {
  matte_f(mat.matte, f);
  f *= mat.matte_pdf;
}

__HD__ void alloy_f(
    const AlloyParam& mat,
    const Vector& wo,
    const Normal& N,
    Spectrum& f) {

  const float c  = 1.f - Dot(wo, N);
  const float Re = mat.R0 + (1.f - mat.R0) * c * c * c * c * c;
  const float P  = .25f + .5f * Re;

  f.r = mat.diff.r * INV_PI;
  f.g = mat.diff.g * INV_PI;
  f.b = mat.diff.b * INV_PI;

  f *= (1.f - Re) / (1.f - P);
}

template<class T> __host__ __device__ void my_atomic_add(T* var, T inc) {
#ifdef __CUDA_ARCH__
  atomicAdd(var, inc);
#else
  __sync_fetch_and_add(var, inc);
#endif
}


__HD__ void add_flux(
    const unsigned*  hash_grid,
    const unsigned*  hash_grid_lengths,
    const unsigned*  hash_grid_indexes,
    const float      hash_grid_inv_cell_size,

    const BBox& bbox,
    const PtrFreeScene* const scene,
    const Point& hit_point,
    const Normal& shade_N,
    const Vector& wi,
    const Spectrum& photon_flux,
    const float photon_radius2,
    HitPointPosition* const hit_points_info,
    HitPointRadiance* const hit_points,
    const unsigned hit_points_count) {

  const Vector hh = (hit_point - bbox.pMin) * hash_grid_inv_cell_size;

  const int ix = abs(int(hh.x));
  const int iy = abs(int(hh.y));
  const int iz = abs(int(hh.z));

  unsigned grid_index = hash(ix, iy, iz, hit_points_count);
  unsigned length = hash_grid_lengths[grid_index];

  if (length > 0) {
    unsigned local_list = hash_grid_indexes[grid_index];
    for(unsigned i = local_list; i < local_list + length; ++i) {
      unsigned hit_point_index = hash_grid[i];
      HitPointPosition& ihp = hit_points_info[hit_point_index];
      HitPointRadiance& hp = hit_points[hit_point_index];

      if (DistanceSquared(ihp.position, hit_point) > photon_radius2 || Dot(ihp.normal, wi) <= 0.0001f)
        continue;

      Spectrum f;

      Material& hit_point_mat = scene->materials[ihp.material_ss];
      switch(hit_point_mat.type) {
        case MAT_MATTE: matte_f(hit_point_mat.param.matte, f);
                        break;
        case MAT_MATTEMIRROR: matte_mirror_f(hit_point_mat.param.matte_mirror, f);
                              break;
        case MAT_MATTEMETAL:  matte_metal_f(hit_point_mat.param.matte_metal, f);
                              break;
        case MAT_ALLOY: alloy_f(hit_point_mat.param.alloy, ihp.wo, shade_N, f);
                        break;
      }

      Spectrum flux = photon_flux * AbsDot(shade_N, wi) * ihp.throughput * f;

#ifdef __CUDA_ARCH__
      my_atomic_add(&hp.reflected_flux.r, flux.r);
      my_atomic_add(&hp.reflected_flux.g, flux.g);
      my_atomic_add(&hp.reflected_flux.b, flux.b);
#else
#pragma omp critical
      {
        hp.reflected_flux = hp.reflected_flux + flux;
      }
#endif
    }
  }
}

__HD__ unsigned hash(const int ix, const int iy, const int iz, unsigned size) {
  return (unsigned) ((ix * 73856093) ^ (iy * 19349663) ^ (iz * 83492791)) % size;
}

}

} }

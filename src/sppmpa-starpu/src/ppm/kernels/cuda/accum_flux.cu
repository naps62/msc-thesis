#include "hip/hip_runtime.h"
#include "ppm/kernels/codelets.h"
using namespace ppm::kernels::codelets;

#include "ppm/kernels/helpers.cuh"
#include "utils/config.h"
#include "ppm/ptrfreescene.h"
#include "utils/random.h"
#include "ppm/types.h"
using ppm::PtrFreeScene;
using ppm::EyePath;

#include <starpu.h>
#include <cstdio>
#include <cstddef>

namespace ppm { namespace kernels { namespace cuda {

void __global__ accum_flux_impl(
    const HitPointPosition* const hit_points_info,
    HitPointRadiance* const hit_points,
    const unsigned size,
    const float alpha,
    const unsigned photons_traced,
    const float* current_photon_radius2) {

  const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size)
    return;

  const float radius2 = *current_photon_radius2;

  const HitPointPosition& hpi = hit_points_info[i];
  HitPointRadiance& hp = hit_points[i];

  switch (hpi.type) {
    case CONSTANT_COLOR:
      hp.radiance = hpi.throughput;
      break;
    case SURFACE:
      break;
    default:
      assert(false);
  }

  const double k = 1.0 / (M_PI * radius2 * photons_traced);
  hp.radiance = hp.radiance + hp.reflected_flux * k;

}


void accum_flux(void* buffers[], void* args_orig) {

  // cl_args
  starpu_args args;
  float alpha;
  unsigned photons_traced;
  starpu_codelet_unpack_args(args_orig, &args, &alpha, &photons_traced);

  // buffers
  const HitPointPosition* const hit_points_info = (const HitPointPosition*)STARPU_VECTOR_GET_PTR(buffers[0]);
        HitPointRadiance*           const hit_points      = (HitPointRadiance*)STARPU_VECTOR_GET_PTR(buffers[1]);
  const unsigned size = STARPU_VECTOR_GET_NX(buffers[0]);

  const float* const photon_radius2 = (const float*)STARPU_VARIABLE_GET_PTR(buffers[2]);

  const unsigned threads_per_block = args.config->cuda_block_size;
  const unsigned n_blocks          = std::ceil(size / (float)threads_per_block);

  printf("accum\n");
  accum_flux_impl
  <<<n_blocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>
   (hit_points_info,
    hit_points,
    size,
    alpha,
    photons_traced,
    photon_radius2);

  hipStreamSynchronize(starpu_cuda_get_local_stream());
  CUDA_SAFE(hipGetLastError());
  printf("accum\n");
}

} } }

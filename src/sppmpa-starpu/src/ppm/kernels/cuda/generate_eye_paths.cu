#include "hip/hip_runtime.h"
#include "ppm/kernels/codelets.h"
using namespace ppm::kernels::codelets;

#include "ppm/kernels/helpers.cuh"

#include "utils/config.h"
#include "ppm/ptrfreescene.h"
#include "utils/random.h"
#include "ppm/types.h"
using ppm::PtrFreeScene;
using ppm::EyePath;

#include <starpu.h>
#include <cstdio>
#include <cstddef>

namespace ppm { namespace kernels { namespace cuda {

void __global__ generate_eye_paths_impl(
    EyePath* const eye_paths, // const unsigned eye_path_count,
    Seed* const seed_buffer,  // const unsigned seed_buffer_count,
    const unsigned width,
    const unsigned height,
    const PtrFreeScene* scene) {

  const unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  const unsigned x = index / width; // TODO check this
  const unsigned y = index % width;

  if (index >= width * height)
    return;

  EyePath& eye_path = eye_paths[index];

  eye_path = EyePath();
  eye_path.scr_y = y + (floatRNG(seed_buffer[index])) - 0.5f;
  eye_path.scr_x = x + (floatRNG(seed_buffer[index])) - 0.5f;

  float u0 = floatRNG(seed_buffer[index]);
  float u1 = floatRNG(seed_buffer[index]);
  float u2 = floatRNG(seed_buffer[index]);

  eye_path.ray = helpers::generate_ray(eye_path.scr_x, eye_path.scr_y, width, height, u0, u1, u2, scene->camera);

  eye_path.done = false;
  eye_path.sample_index = index;
}


void generate_eye_paths(void* buffers[], void* args_orig) {
  // cl_args
  const starpu_args args;
  starpu_codelet_unpack_args(args_orig, &args);

  // buffers
  // eye_paths
  EyePath* const eye_paths = (EyePath*)STARPU_VECTOR_GET_PTR(buffers[0]);
  //const unsigned eye_path_count = STARPU_VECTOR_GET_NX(buffers[0]);
  // seeds
  Seed* const seed_buffer  = (Seed*)STARPU_VECTOR_GET_PTR(buffers[1]);
  //const unsigned seed_buffer_count = STARPU_VECTOR_GET_NX(buffers[1]);

  const unsigned width = args.gpu_config->width;
  const unsigned height = args.gpu_config->height;
  const unsigned threads_per_block = args.gpu_config->cuda_block_size;
  const unsigned n_blocks          = width * height / args.gpu_config->cuda_block_size;

  generate_eye_paths_impl<<<n_blocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>
   (eye_paths,   // eye_path_count,
    seed_buffer, // seed_buffer_count,
    width,
    height,
    args.gpu_scene);


}

} } }
